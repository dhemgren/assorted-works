#include "hip/hip_runtime.h"
#include "main.h"

int n, m;

int *v;
int *e;
int *w;

int *c;
int *f;
int *u;

int mssp;

int block_dim_minimum1;
int num_blocks_minimum1;

int block_dim_relax_update;
int num_blocks_relax_update;

// Pointers to device memory
int *vd;
int *ed;
int *wd;

int *cd;
int *fd;
int *ud;

__global__ void relax_helper(int *v, int *e, int *w, int *c, int *f, int *u, int n) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index > n-1)
        return;

    if (f[index]) {
        int start = v[index];
        int end = v[index+1];
        for (int j = start; j < end; j++) {
            int to = e[j];
            if (u[to]) {
                // Writes to c[to].
                atomicMin(&(c[to]), c[index] + w[j]);
            }
        }
    }
}

// relax_F
void relax(void) {
    // 1 thread block, n threads per block.
    relax_helper<<<num_blocks_relax_update, block_dim_relax_update>>>
        (vd, ed, wd, cd, fd, ud, n);
}

__global__ void minimum1_helper(int *c, const int *u, int *minimums, int n) {
    int out_of_bounds = false;
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= n)
        out_of_bounds = true;

    int thid = threadIdx.x;
    int *sdata = c + (blockIdx.x * blockDim.x);

    int i = blockIdx.x * (2 * blockDim.x) + threadIdx.x;
    int j = i + blockDim.x;

    int data1;
    int data2;

    if (i >= n)
        data1 = INT_MAX;
    if (i < n)
        data1 = u[i] ? c[i] : INT_MAX;

    if (j >= n)
        data2 = INT_MAX;
    if (j < n)
        data2 = u[j] ? c[j] : INT_MAX;

    if (!out_of_bounds)
        sdata[thid] = min(data1, data2);
    __syncthreads();
    for (int s = blockDim.x/2; s > 0; s>>=1) { // Dela s på 2 efter varje iteration
        if (thid < s && !out_of_bounds && index + s < n) {
            sdata[thid] = min(sdata[thid], sdata[thid+s]);
        }
        __syncthreads();
    }
    if (thid == 0 && !out_of_bounds) {
        minimums[blockIdx.x] = sdata[0];
    }
}

int minimum1(void) {
    //int num_threads_total = nearest_power_of_2(n) / 2;
    //int num_blocks = 1; // TODO: smartare val
    //int num_threads_per_block = num_threads_total / num_blocks;

    int *minimumsd;
    hipMalloc((void **) &minimumsd, num_blocks_minimum1 * sizeof(int));

    int *cd_working_copy;
    hipMalloc((void **) &cd_working_copy, n * sizeof(int));
    hipMemcpy(cd_working_copy, cd, n * sizeof(int), hipMemcpyDeviceToDevice);

    minimum1_helper<<<num_blocks_minimum1, block_dim_minimum1>>>(cd_working_copy, ud, minimumsd, n);

    int *minimums = (int *) malloc(num_blocks_minimum1 * sizeof(int));
    hipMemcpy(minimums, minimumsd, num_blocks_minimum1 * sizeof(int), hipMemcpyDeviceToHost);

    int res = INT_MAX;
    /* printf("MINIMUMS\n"); */
    for (int i = 0; i < num_blocks_minimum1; i++) {
        /* printf("%d\n", minimums[i]); */
        if (minimums[i] < res)
            res = minimums[i];
    }
    free(minimums);
    hipFree(minimumsd);
    hipFree(cd_working_copy);
    return res;
}

__global__ void update_helper(int *c, int *f, int *u, int mssp, int n) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index > n-1)
        return;

    f[index] = false;
    if (c[index] == mssp) {
        u[index] = false;
        f[index] = true;
    }
}

void update(void) {
    update_helper<<<num_blocks_relax_update, block_dim_relax_update>>>
        (cd, fd, ud, mssp, n);
}

void da2cf(void) {
    host_to_device();

    mssp = 0;
    int mssp_cu = 0;

    while (true) {
        relax();
        device_to_host(); // In order to calculate new mssp on host. To remove later.
        
        mssp = minimum_sequential();
        mssp_cu = minimum1();

        /* printf("%d %d\n", mssp, mssp_cu); */

        if (mssp_cu == INT_MAX)
            break;

        update();
    }

    device_to_host();
    print_res();
}

int main(void) {
    init();
    da2cf();
    free_memory();
}
