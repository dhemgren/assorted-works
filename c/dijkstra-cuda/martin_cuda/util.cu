#include "main.h"

void print_c_u(void) {
    printf("\n\nu:\n");
    for(int i = 0; i < n; i++){
        printf("u[%d]=%d\n", i, u[i]);
    }

    printf("\n\nc:\n");
    for(int i = 0; i < n; i++){
        printf("c[%d]=%d\n", i, c[i]);
    }
}

void print_res(void) {
    for (int i = 0; i < n; i++) {
        printf("%d %d\n", i, c[i]);
    }
}

int nearest_power_of_2(int arg) {
    unsigned int x = (unsigned int) arg;
    x--;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    x++; 
    return x;
}

void host_to_device(void) {
    hipMemcpy(vd, v, (n+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ed, e, m * sizeof(int),     hipMemcpyHostToDevice);
    hipMemcpy(wd, w, m * sizeof(int),     hipMemcpyHostToDevice);

    hipMemcpy(cd, c, n * sizeof(int),     hipMemcpyHostToDevice);
    hipMemcpy(fd, f, n * sizeof(int),     hipMemcpyHostToDevice);
    hipMemcpy(ud, u, n * sizeof(int),     hipMemcpyHostToDevice);
}

void device_to_host(void) {
    hipMemcpy(v, vd, (n+1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(e, ed, m * sizeof(int),     hipMemcpyDeviceToHost);
    hipMemcpy(w, wd, m * sizeof(int),     hipMemcpyDeviceToHost);

    hipMemcpy(c, cd, n * sizeof(int),     hipMemcpyDeviceToHost);
    hipMemcpy(f, fd, n * sizeof(int),     hipMemcpyDeviceToHost);
    hipMemcpy(u, ud, n * sizeof(int),     hipMemcpyDeviceToHost);
}

// Precondition: n and m are set
void allocate_memory(void) {
    v = (int *) malloc((n+1) * sizeof(int));
    e = (int *) malloc(m * sizeof(int));
    w = (int *) malloc(m * sizeof(int));
    c = (int *) malloc(n * sizeof(int));
    f = (int *) malloc(n * sizeof(int));
    u = (int *) malloc(n * sizeof(int));

    assert(hipMalloc((void **) &vd, (n+1) * sizeof(int)) == hipSuccess);
    assert(hipMalloc((void **) &ed, m * sizeof(int)) == hipSuccess);
    assert(hipMalloc((void **) &wd, m * sizeof(int)) == hipSuccess);
    assert(hipMalloc((void **) &cd, n * sizeof(int)) == hipSuccess);
    assert(hipMalloc((void **) &fd, n * sizeof(int)) == hipSuccess);
    assert(hipMalloc((void **) &ud, n * sizeof(int)) == hipSuccess);
}

// Precondition: allocate_memory has ran
void free_memory(void) {
    hipFree(vd);
    hipFree(ed);
    hipFree(wd);
    hipFree(cd);
    hipFree(fd);
    hipFree(ud);

    free(v);
    free(e);
    free(w);
    free(c);
    free(f);
    free(u);
}
